#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cmath>
#include <time.h> 
#define BLOCK_SIZE  16          // submatrix size

__global__ void matMultCuda(float* a, float* b, int n, float* c)
{
    int bx = blockIdx.x;        // block index
    int by = blockIdx.y;

    int tx = threadIdx.x;       // thread index
    int ty = threadIdx.y;

    // Index of the first sub-matrix of A processed by the block
    int aBegin = n * BLOCK_SIZE * by;
    int aEnd = aBegin + n - 1;
    // Step size used to iterate through the sub-matrices of A
    int aStep = BLOCK_SIZE;
    // Index of the first sub-matrix of B processed by the block
    int bBegin = BLOCK_SIZE * bx;
    // Step size used to iterate through the sub-matrices of B
    int bStep = BLOCK_SIZE * n;
    float sum = 0.0f;           // computed subelement

    for (int ia = aBegin, ib = bBegin; ia <= aEnd; ia += aStep, ib += bStep)
    {
        // Shared memory for the sub-matrix of A
        __shared__ float as[BLOCK_SIZE][BLOCK_SIZE];
        // Shared memory for the sub-matrix of B
        __shared__ float bs[BLOCK_SIZE][BLOCK_SIZE];

        // Load the matrices from global memory to shared memory;
        as[ty][tx] = a[ia + n * ty + tx];
        bs[ty][tx] = b[ib + n * ty + tx];

        __syncthreads();    // Synchronize to make sure the matrices are loaded

                            // Multiply the two matrices together;
        for (int k = 0; k < BLOCK_SIZE; k++)
            sum += as[ty][k] * bs[k][tx];

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write the block sub-matrix to global memory;
    // each thread writes one element
    int ic = n * BLOCK_SIZE * by + BLOCK_SIZE * bx;

    c[ic + n * ty + tx] = sum;
}

double calcCuda(float* a, float* b, float* c, int N, bool flag)
{
    clock_t start2 = clock();
    int numBytes = N * N * sizeof(float);
    // allocate device memory
    float* adev = NULL;
    float* bdev = NULL;
    float* cdev = NULL;

    hipMalloc((void**)&adev, numBytes);
    hipMalloc((void**)&bdev, numBytes);
    hipMalloc((void**)&cdev, numBytes);

    // set kernel launch configuration
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocks(N / threads.x, N / threads.y);

    // create cuda event handles
    hipEvent_t start, stop;
    float gpuTime = 0.0f;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    // asynchronously issue work to the GPU (all to stream 0)
    hipEventRecord(start, 0);
    hipMemcpy(adev, a, numBytes, hipMemcpyHostToDevice);
    hipMemcpy(bdev, b, numBytes, hipMemcpyHostToDevice);

    matMultCuda << < blocks, threads >> > (adev, bdev, N, cdev);

    hipMemcpy(c, cdev, numBytes, hipMemcpyDeviceToHost);
    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpuTime, start, stop);

    // print the events gpu times
    if (flag)
        printf("Time spent executing by the GPU events: %.2f millseconds\n", gpuTime);

    // release resources
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(adev);
    hipFree(bdev);
    hipFree(cdev);
    clock_t end2 = clock();
    double millseconds2 = (double)(end2 - start2);
    // print the gpu times
    if (flag)
        printf("Time spent executing by the GPU: %.2f millseconds\n", millseconds2);
    return millseconds2;
}

double calcCPU(float* a, float* b, float* c, int N)
{
    clock_t start3 = clock();
    for (int i = 0; i < N; ++i)
    {
        for (int j = 0; j < N; ++j)
        {
            c[i * N + j] = 0;
            for (int k = 0; k < N; ++k)
                c[i * N + j] += a[i * N + k] * b[k * N + j];
        }
    }
    clock_t end3 = clock();
    double millseconds = (double)(end3 - start3);
    // print the cpu times
    printf("Time spent executing by the CPU: %.2f millseconds\n", millseconds);
    return millseconds;
}

int main(int argc, char* argv[])
{
    float* a = new float[64 * 64];
    float* b = new float[64 * 64];
    float* cpu = new float[64 * 64];
    float* gpu = new float[64 * 64];
    //Run to initialize cuda
    calcCuda(a, b, gpu, 64, false);
    //Main
    for (int i = 6; i < 12; i++)
    {
        int N = pow(2, i);       // matrix size is N*N
        printf("Experiment for matrix size: %u \n", N);
        // allocate host memory
        float* a = new float[N * N];
        float* b = new float[N * N];
        float* cpu = new float[N * N];
        float* gpu = new float[N * N];

        for (int i = 0; i < N; i++)
            for (int j = 0; j < N; j++)
            {
                a[i * N + j] = rand() % 100;
                b[i * N + j] = rand() % 100;
            }
        printf("Acceleration factor: %.2f \n", calcCPU(a, b, cpu, N) / calcCuda(a, b, gpu, N, true));
        bool rel = true;
        for (int i = 0; i < N; i++)
            for (int j = 0; j < N; j++)
            {
                if (cpu[i * N + j] != gpu[i * N + j]) { rel = false; break; }
            }
        printf("Relevance: %s \n", rel ? "true" : "false");
        delete a;
        delete b;
        delete cpu;
        delete gpu;
    }
    return 0;
}